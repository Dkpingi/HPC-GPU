#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"
__device__ long int dummyFlag;
__global__
void
WaitKernel(long int waitTime)
{


clock_t clockstart = clock();
long int timeDiff = clock() - clockstart;

//printf("timeDiff: %li \n waitTime: %li \n",timeDiff,waitTime);
while (timeDiff < waitTime)
{
//printf("(insideloop)timeDiff: %li \n waitTime: %li \n",timeDiff,waitTime);

timeDiff = clock() - clockstart;
//printf("(after new calc)timeDiff: %li \n waitTime: %li \n",timeDiff,waitTime);

if (threadIdx.x == 1000000)
{
	dummyFlag = timeDiff;
}
}
//printf("(endofkernel)timeDiff: %li \n waitTime: %li \n",timeDiff,waitTime);
}

int
main(int argc,char **argv)
{
    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    int clockrate = props.clockRate;
    printf("clockrate: %d\n",clockrate); 
    long int waitTimeCl = atoi(argv[1]);
    const int cIterations = 1e5;
    printf( "Measuring Asynchronous launch time... " ); fflush( stdout );
    //printf( "dummyFlag = %.5g\n",dummyFlag ); fflush( stdout );
    chTimerTimestamp start, stop;

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        WaitKernel<<<1,1>>>(waitTimeCl);
    }
    hipDeviceSynchronize();
    chTimerGetTime( &stop );

    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;
	double AsyncTime = usPerLaunch;

        printf( "%.2f us\n", usPerLaunch );
	printf( "%li clocks per sec\n", CLOCKS_PER_SEC );
	FILE *fp;
	fp = fopen("BreakEven.txt","a");
	fprintf (fp, "%li %.5g\n", waitTimeCl, AsyncTime );
	fclose(fp);
    }

    return 0;
}
