/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : TODO
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <hip/hip_runtime.h>
#include <cstdio>
// Simple reduction kernel in global memory
// Unoptimized version
// Works, same as CPU
__global__ void
reduction_Kernel(int numElements, float *d_out, float *d_in)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	if (myId < numElements)
	{
		// do reduction in global mem
		// Go down from 1024 ... 512 ... 256 ... 128 ....... 1
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
		{
			if (tid < s)
			{
				d_in[myId] += d_in[myId + s];
			}
			__syncthreads();
		}

		// At the end, only thread 0 is out
		if (tid == 0)
		{
			d_out[blockIdx.x] = d_in[myId];
		}
	}
}

// Simple reduction wrapper
void reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float *dataIn, float *dataOut)
{
	float *rs; // tmp
	hipMalloc(&rs, gridSize.x * sizeof(float));

	reduction_Kernel<<<gridSize, blockSize>>>(numElements, rs, dataIn);
	hipDeviceSynchronize();
	// Only start 1 block
	//reduction_Kernel<<<gridSize, blockSize>>>(numElements, dataOut, rs);
	reduction_Kernel<<<1, gridSize>>>(numElements, dataOut, rs);

	hipFree(rs);
}

// Optimized reduction kernel
// We used technique:
// SEQUENTIAL ADDRESSING NONDIVERGENT
__global__ void
opt_reduction_Kernel(int numElements, float *dataIn, float *dataOut)
{
	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];

	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	if (myId < numElements)
	{
		// load shared mem from global mem
		sdata[tid] = dataIn[myId];
		__syncthreads(); // make sure entire block is loaded!

		// do reduction in shared mem
		// Basically same as normal version
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
		{
			if (tid < s)
			{
				sdata[tid] += sdata[tid + s];
			}
			__syncthreads(); // make sure all adds at one stage are done!
		}

		// only thread 0 writes result for this block back to global mem
		if (tid == 0)
		{
			dataOut[blockIdx.x] = sdata[0];
		}
	}
}

void opt_reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float *dataIn, float *dataOut)
{
	float *rs; // tmp
	hipMalloc(&rs, gridSize.x * sizeof(float));
	// Calculate shared memory size
	opt_reduction_Kernel<<<gridSize, blockSize, blockSize.x * sizeof(float)>>>(numElements, dataIn, rs);
	hipDeviceSynchronize();
	// Only start 1 block (exc)
	opt_reduction_Kernel<<<1, gridSize, gridSize.x * sizeof(float)>>>(numElements, rs, dataOut);

	hipFree(rs);

}
