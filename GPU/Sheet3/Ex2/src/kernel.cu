#include "hip/hip_runtime.h"
/*************************************************************************************************
 *
 *        Computer Engineering Group, Heidelberg University - GPU Computing Exercise 03
 *
 *                           Group : TBD
 *
 *                            File : main.cu
 *
 *                         Purpose : Memory Operations Benchmark
 *
 *************************************************************************************************/

//
// Kernels
//

__global__ void 
globalMemCoalescedKernel(int *src_array,int *copy_array, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int m = floorf(id/size);
    for (int i = 1; i < m + 1; i++)
	{
		if (i*id < size)
		{
		copy_array[i*id] = src_array[i*id];
		}
		syncthreads();
	}
    
}

void 
globalMemCoalescedKernel_Wrapper(dim3 gridDim, dim3 blockDim,int *src_array,int *copy_array, int size) {
	globalMemCoalescedKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>(src_array,copy_array,size);
}

__global__ void 
globalMemStrideKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemStrideKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemStrideKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

__global__ void 
globalMemOffsetKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemOffsetKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemOffsetKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

