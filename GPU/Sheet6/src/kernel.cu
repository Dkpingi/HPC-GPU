/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : TODO
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <hip/hip_runtime.h>
#include <cstdio>
//
// Reduction_Kernel
//
__global__ void
reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float blockSum[];
	int tid = threadIdx.x;
	if (elementId < numElements)
	{
	        blockSum[tid] = dataIn[elementId];
	} 
	else
	{
		blockSum[tid] = 0.0;
	}
	__syncthreads();
	
	for(int s = 1;s< blockDim.x; s*= 2)
	{
		if(tid%(2*s) == 0)
		{
			blockSum[tid] += blockSum[tid + s];
		}
		__syncthreads();
	}
		
	if(tid == 0)
		*(dataOut + blockIdx.x) = blockSum[0];

}

void reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) {
	float* buffer = NULL;
	hipMalloc(&buffer,gridSize.x*sizeof(float));
	reduction_Kernel<<< gridSize, blockSize, blockSize.x*sizeof(float) >>>(numElements, dataIn, buffer);
	hipDeviceSynchronize();

	unsigned int nthreads = 1;
	while(nthreads < gridSize.x)
	{
		nthreads*=2;
	}
	reduction_Kernel<<< 1, nthreads, nthreads*sizeof(float) >>>(gridSize.x, buffer, dataOut);


}


__global__ void
opt_reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
	int elementId = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	extern __shared__ float blockSum[];
	int tid = threadIdx.x;
	if (elementId < numElements)
	{
		blockSum[tid] = dataIn[elementId] + dataIn[elementId + blockDim.x];
	}
	else
	{
		blockSum[tid] = 0.0;
	}

	__syncthreads();	
	for(unsigned int s = blockDim.x/2;s>32; s >>= 1)
	{
		if(tid < s)
		{
			blockSum[tid] += blockSum[tid + s];
		}
		__syncthreads();
	}
 	if( tid < 32 && blockDim.x >= 64) blockSum[tid] += blockSum[tid + 32];  
 	if ( tid < 16 && blockDim.x >= 32) blockSum[tid] += blockSum[tid + 16];  
 	if ( tid <  8 && blockDim.x >= 16) blockSum[tid] += blockSum[tid + 8];   
 	if ( tid <  4 && blockDim.x >=  8) blockSum[tid] += blockSum[tid + 4];   
 	if ( tid <  2 && blockDim.x >=  4) blockSum[tid] += blockSum[tid + 2];   
 	if ( tid <  1 && blockDim.x >=  2) blockSum[tid] += blockSum[tid + 1];
	
			
	if(tid == 0)
		*(dataOut + blockIdx.x) = blockSum[0];
	
}

void opt_reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) {
	float* buffer = NULL;
	hipMalloc(&buffer,gridSize.x*sizeof(float));
	blockSize.x /= 2;
	opt_reduction_Kernel<<< gridSize, blockSize, blockSize.x*sizeof(float) >>>(numElements, dataIn, buffer);
	hipDeviceSynchronize();
	unsigned int nthreads = 1;
	while(nthreads < gridSize.x/2)
	{
		nthreads*=2;
	}
	opt_reduction_Kernel<<< 1, nthreads, nthreads*sizeof(float) >>>(gridSize.x, buffer, dataOut);

}
