/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : TODO
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <hip/hip_runtime.h>
#include <cstdio>
//
// Reduction_Kernel
//
__global__ void
reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float blockSum[];
	//printf("Thread %d start", threadIdx.x);
	if (elementId < numElements)
	{
		//printf("ThreadId: %d\n", threadIdx.x);
		//printf("ElementID: %d\n", elementId);
		//printf("NumElements: %d\n", numElements);
	        blockSum[threadIdx.x] = dataIn[elementId];
		//printf("thread just wrote: %d\n", threadIdx.x);
	} 
	else
	{
		//printf("%d\n", threadIdx.x);
		//printf("%d\n", elementId);
		//printf("%d\n", numElements);
		blockSum[threadIdx.x] = 0.0;
	}
	__syncthreads();
	
	for(int s = 1;s< blockDim.x; s*= 2)
	{
		//printf("4\n");
		int index = 2*s*threadIdx.x;
		if(index < blockDim.x)
		{
			//printf("5\n");
			blockSum[index] += blockSum[index + s];
		}
		__syncthreads();
	}
		
	if(threadIdx.x == 0)
		*(dataOut + blockIdx.x) = blockSum[0];
	
	//printf("6\n");
}

void reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) {
	float* buffer = NULL;
	hipMalloc(&buffer,gridSize.x*sizeof(float));
	//printf("7\n");
	reduction_Kernel<<< gridSize, blockSize, blockSize.x*sizeof(float) >>>(numElements, dataIn, buffer);
	unsigned int nthreads = 1;
	while(nthreads < gridSize.x)
	{
		nthreads*=2;
	}
	hipDeviceSynchronize();
	//printf("8\n");
	reduction_Kernel<<< 1, nthreads, nthreads*sizeof(float) >>>(gridSize.x, buffer, dataOut);
	//printf("9\n");
	hipFree(buffer);

}
