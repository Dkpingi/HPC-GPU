#include "hip/hip_runtime.h"
/******************************************************************************
 *
 *           XXXII Heidelberg Physics Graduate Days - GPU Computing
 *
 *                 Gruppe : TODO
 *
 *                   File : main.cu
 *
 *                Purpose : n-Body Computation
 *
 ******************************************************************************/

#include <cmath>
#include <ctime>
#include <iostream>
#include <cstdlib>
#include <chCommandLine.h>
#include <chTimer.hpp>
#include <cstdio>
#include <iomanip>
#include <fstream>

const static int DEFAULT_NUM_ELEMENTS   = 1024;
const static int DEFAULT_NUM_ITERATIONS =    5;
const static int DEFAULT_BLOCK_DIM      =  128;

const static float TIMESTEP =      1e-6; // s
const static float GAMMA    = 6.673e-11; // (Nm^2)/(kg^2)

//
// Structures
//
// Use a SOA (Structure of Arrays)
//
struct Body_t {
	float4* posMass;  /* x = x */
	                  /* y = y */
	                  /* z = z */
	                  /* w = Mass */
	float3* velocity; /* x = v_x*/
	                  /* y = v_y */
	                  /* z= v_z */
	
	Body_t(): posMass(NULL), velocity(NULL) {}
	};

//
// Function Prototypes
//
void printHelp(char *);
void printElement(Body_t, int, int);

//
// Device Functions
//

//
// Calculate the Distance of two points
//
__device__ float
getDistance(float4 a, float4 b)
{
	// TODO: Calculate distance of two particles
	return norm3df(a.x - b.x,a.y - b.y,a.z - b.z);
}

//
// Calculate the forces between two bodies
//
__device__ void
bodyBodyInteraction(float4 bodyA, float4 bodyB, float3& force)
{
	float distance = getDistance(bodyA, bodyB);

	if (distance==0)
	{
		return;
	}
	else
	{	
		float invDistanceCubed = 1/(distance*distance*distance);
		force.x = GAMMA*bodyA.w*bodyB.w*(bodyB.x - bodyA.x)*invDistanceCubed;
		force.y = GAMMA*bodyA.w*bodyB.w*(bodyB.y - bodyA.y)*invDistanceCubed;
		force.z = GAMMA*bodyA.w*bodyB.w*(bodyB.z - bodyA.z)*invDistanceCubed;
	}
	// TODO: Calc Force
}

__device__ void
bodyBodyInteraction_opt(float4 bodyA, float4 bodyB, float3& accel)
{
	float dx = (bodyB.x - bodyA.x);
	float dy = (bodyB.y - bodyA.y);
	float dz = (bodyB.z - bodyA.z);
	float InvDistance = rsqrtf(dx*dx + dy*dy + dz*dz);
	float CinvDistanceCubed = bodyB.w*(InvDistance*InvDistance*InvDistance);
	accel.x += dx*CinvDistanceCubed;
	accel.y += dy*CinvDistanceCubed;
	accel.z += dz*CinvDistanceCubed;
	// TODO: Calc Force
}

//
// Calculate the new velocity of one particle
//
__device__ void
calculateSpeed(float mass, float3& currentSpeed, float3 force)
{	
	currentSpeed.x += (force.x/mass)*TIMESTEP;
	currentSpeed.y += (force.y/mass)*TIMESTEP;
	currentSpeed.z += (force.z/mass)*TIMESTEP;
	
	// TODO: Calculate the new velocity of a particle
}

__device__ void
calculateSpeed_opt(float3& currentSpeed, float3 accel)
{
	currentSpeed.x += accel.x*TIMESTEP;
	currentSpeed.y += accel.y*TIMESTEP;
	currentSpeed.z += accel.z*TIMESTEP;
	
	// TODO: Calculate the new velocity of a particle
}

//
// n-Body Kernel for the speed calculation
//
__global__ void
simpleNbody_Kernel(int numElements, float4* bodyPos, float3* bodySpeed)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	
	for(int i = elementId; i<numElements; i+= gridDim.x*blockDim.x)
	{
		float4 elementPosMass = bodyPos[i];
		float3 elementSpeed = bodySpeed[i];
		float3 elementForce = make_float3(0,0,0);
		for (int j = 0; j < numElements; j++) {
			float3 tempForce = make_float3(0,0,0);
			bodyBodyInteraction(elementPosMass, bodyPos[j], tempForce);
			elementForce.x += tempForce.x;
			elementForce.y += tempForce.y;
			elementForce.z += tempForce.z;
		}

		calculateSpeed(elementPosMass.w, elementSpeed, elementForce);

		bodySpeed[i] = elementSpeed;
	}
}

__global__ void
sharedNbody_Kernel(const int numElements, const float4* bodyPos, float3* bodySpeed)
{
	extern __shared__ float4 posMass[];
	const int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	const int tid = threadIdx.x;
	for(int i = elementId; i<numElements; i += blockDim.x*gridDim.x)
	{
		const float4 elementbodyPos = bodyPos[i];
		float3 elementAccel = make_float3(0,0,0);
		#pragma unroll 16 // slight performance increase
		for(int j = 0;j<numElements;j+= blockDim.x)
		{
			posMass[tid] = bodyPos[tid + j];
			posMass[tid].w *= GAMMA;
			__syncthreads();
			if(abs(j-i) < blockDim.x)
			{
				#pragma unroll 32 // significant performance increase 
				for(int k = 0; k<blockDim.x; k++)
				{
					if(i != j + k)
					{
						bodyBodyInteraction_opt(elementbodyPos, posMass[k], elementAccel);
					}
				}
			}
			else
			{
				#pragma unroll 32
				for(int k = 0; k<blockDim.x; k++)
				{
					bodyBodyInteraction_opt(elementbodyPos, posMass[k], elementAccel);
				}

			}
			__syncthreads();
		}
		calculateSpeed_opt(bodySpeed[i], elementAccel);
	}		
}

//
// n-Body Kernel to update the position
// Neended to prevent write-after-read-hazards
//
__global__ void
updatePosition_Kernel(int numElements, float4* bodyPos, float3* bodySpeed)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i = elementId; i<numElements; i+= gridDim.x*blockDim.x)
	{
		bodyPos[i].x += bodySpeed[i].x*TIMESTEP;
		bodyPos[i].y += bodySpeed[i].y*TIMESTEP;
		bodyPos[i].z += bodySpeed[i].z*TIMESTEP;
	}
}

//
// Main
//
int
main(int argc, char * argv[])
{
	bool showHelp = chCommandLineGetBool("h", argc, argv);
	if (!showHelp) {
		showHelp = chCommandLineGetBool("help", argc, argv);
	}

	if (showHelp) {
		printHelp(argv[0]);
		exit(0);
	}

	std::cout << "***" << std::endl
			  << "*** Starting ..." << std::endl
			  << "***" << std::endl;

	ChTimer memCpyH2DTimer, memCpyD2HTimer;
	ChTimer kernelTimer;

	//
	// Allocate Memory
	//
	int numElements = 0;
	chCommandLineGet<int>(&numElements, "s", argc, argv);
	chCommandLineGet<int>(&numElements, "size", argc, argv);
	numElements = numElements != 0 ?
			numElements : DEFAULT_NUM_ELEMENTS;
	//
	// Host Memory
	//
	bool pinnedMemory = chCommandLineGetBool("p", argc, argv);
	if (!pinnedMemory) {
		pinnedMemory = chCommandLineGetBool("pinned-memory",argc,argv);
	}

	Body_t h_particles;
	if (!pinnedMemory) {
		// Pageable
		h_particles.posMass = static_cast<float4*>
				(malloc(static_cast<size_t>
				(numElements * sizeof(*(h_particles.posMass)))));
		h_particles.velocity = static_cast<float3*>
				(malloc(static_cast<size_t>
				(numElements * sizeof(*(h_particles.velocity)))));
	} else {
		// Pinned
		hipHostMalloc(&(h_particles.posMass), 
				static_cast<size_t>
				(numElements * sizeof(*(h_particles.posMass))));
		hipHostMalloc(&(h_particles.velocity), 
				static_cast<size_t>
				(numElements * sizeof(*(h_particles.velocity))));
	}

	// Init Particles
//	srand(static_cast<unsigned>(time(0)));
	srand(0); // Always the same random numbers
	for (int i = 0; i < numElements; i++) {
		h_particles.posMass[i].x = 1e-8*static_cast<float>(rand()); // Modify the random values to
		h_particles.posMass[i].y = 1e-8*static_cast<float>(rand()); // increase the position changes
		h_particles.posMass[i].z = 1e-8*static_cast<float>(rand()); // and the velocity
		h_particles.posMass[i].w =  1e4*static_cast<float>(rand());
		h_particles.velocity[i].x = 0.0f;
		h_particles.velocity[i].y = 0.0f;
		h_particles.velocity[i].z = 0.0f;
	}
	
	//printElement(h_particles, 0, 0);

	// Device Memory
	Body_t d_particles;
	hipMalloc(&(d_particles.posMass), 
			static_cast<size_t>(numElements * sizeof(*(d_particles.posMass))));
	hipMalloc(&(d_particles.velocity), 
			static_cast<size_t>(numElements * sizeof(*(d_particles.velocity))));

	if (h_particles.posMass == NULL || h_particles.velocity == NULL ||
		d_particles.posMass == NULL || d_particles.velocity == NULL) {
		std::cout << "\033[31m***" << std::endl
		          << "*** Error - Memory allocation failed" << std::endl
		          << "***\033[0m" << std::endl;

		exit(-1);
	}

	//
	// Copy Data to the Device
	//
	memCpyH2DTimer.start();

	hipMemcpy(d_particles.posMass, h_particles.posMass, 
			static_cast<size_t>(numElements * sizeof(float4)), 
			hipMemcpyHostToDevice);
	hipMemcpy(d_particles.velocity, h_particles.velocity, 
			static_cast<size_t>(numElements * sizeof(float3)), 
			hipMemcpyHostToDevice);

	memCpyH2DTimer.stop();

	//
	// Get Kernel Launch Parameters
	//
	int blockSize = 0,
	    gridSize = 0,
	    numIterations = 0;

	// Number of Iterations	
	chCommandLineGet<int>(&numIterations,"i", argc, argv);
	chCommandLineGet<int>(&numIterations,"num-iterations", argc, argv);
	numIterations = numIterations != 0 ? numIterations : DEFAULT_NUM_ITERATIONS;

	// Block Dimension / Threads per Block
	chCommandLineGet<int>(&blockSize,"t", argc, argv);
	chCommandLineGet<int>(&blockSize,"threads-per-block", argc, argv);
	blockSize = blockSize != 0 ? blockSize : DEFAULT_BLOCK_DIM;

	if (blockSize > 1024) {
		std::cout << "\033[31m***" << std::endl
		          << "*** Error - The number of threads per block is too big" << std::endl
		          << "***\033[0m" << std::endl;

		exit(-1);
	}

	gridSize = ceil(static_cast<float>(numElements) / static_cast<float>(blockSize));

	dim3 grid_dim = dim3(gridSize);
	dim3 block_dim = dim3(blockSize);

	std::cout << "***" << std::endl;
	std::cout << "*** Grid: " << gridSize << std::endl;
	std::cout << "*** Block: " << blockSize << std::endl;
	std::cout << "***" << std::endl;


	bool optimized = chCommandLineGetBool("opt", argc, argv);
	
	if(!optimized)
	{
	
		kernelTimer.start();

		for (int i = 0; i < numIterations; i ++) {
			simpleNbody_Kernel<<<grid_dim, block_dim>>>(numElements, d_particles.posMass, 
					d_particles.velocity);
			updatePosition_Kernel<<<grid_dim, block_dim>>>(numElements, d_particles.posMass,
					d_particles.velocity);
	
			//hipMemcpy(h_particles.posMass, d_particles.posMass, sizeof(float4), hipMemcpyDeviceToHost);
			//hipMemcpy(h_particles.velocity, d_particles.velocity, sizeof(float3), hipMemcpyDeviceToHost);
			//printElement(h_particles, 0, i+1);
		}
		// Synchronize
		hipDeviceSynchronize();

		// Check for Errors
		hipError_t hipError_t = hipGetLastError();
		if ( hipError_t != hipSuccess ) {
			std::cout << "\033[31m***" << std::endl
			          << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
			          << std::endl
			          << "***\033[0m" << std::endl;

			return -1;
		}

		kernelTimer.stop();

	}
	else
	{
		kernelTimer.start();

		for (int i = 0; i < numIterations; i ++) {
			sharedNbody_Kernel<<<grid_dim, block_dim,block_dim.x*4*sizeof(float)>>>(numElements, 						d_particles.posMass, 
					d_particles.velocity);
			updatePosition_Kernel<<<grid_dim, block_dim>>>(numElements, d_particles.posMass,
					d_particles.velocity);

			//hipMemcpy(h_particles.posMass, d_particles.posMass, sizeof(float4), hipMemcpyDeviceToHost);
			//hipMemcpy(h_particles.velocity, d_particles.velocity, sizeof(float3), hipMemcpyDeviceToHost);
			//printElement(h_particles, 0, i+1);
		}
		//printElement(h_particles, 0, numIterations);
		// Synchronize
		hipDeviceSynchronize();

		// Check for Errors
		hipError_t hipError_t = hipGetLastError();
		if ( hipError_t != hipSuccess ) {
			std::cout << "\033[31m***" << std::endl
			          << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
			          << std::endl
			          << "***\033[0m" << std::endl;
	
			return -1;
		}

		kernelTimer.stop();

	}
	//
	// Copy Back Data
	//
	memCpyD2HTimer.start();
	
	hipMemcpy(h_particles.posMass, d_particles.posMass, 
			static_cast<size_t>(numElements * sizeof(*(h_particles.posMass))), 
			hipMemcpyDeviceToHost);
	hipMemcpy(h_particles.velocity, d_particles.velocity, 
			static_cast<size_t>(numElements * sizeof(*(h_particles.velocity))), 
			hipMemcpyDeviceToHost);

	memCpyD2HTimer.stop();

	printElement(h_particles, 0, numIterations);
	// Free Memory
	if (!pinnedMemory) {
		free(h_particles.posMass);
		free(h_particles.velocity);
	} else {
		hipHostFree(h_particles.posMass);
		hipHostFree(h_particles.velocity);
	}

	hipFree(d_particles.posMass);
	hipFree(d_particles.velocity);
	
	// Print Meassurement Results
    std::cout << "***" << std::endl
	      << "*** Mode: opt = " << optimized << std::endl
              << "*** Results:" << std::endl
              << "***    Num Elements: " << numElements << std::endl
              << "***    Time to Copy to Device: " << 1e3 * memCpyH2DTimer.getTime()
                << " ms" << std::endl
              << "***    Copy Bandwidth: " 
                << 1e-9 * memCpyH2DTimer.getBandwidth(numElements * sizeof(h_particles))
                << " GB/s" << std::endl
              << "***    Time to Copy from Device: " << 1e3 * memCpyD2HTimer.getTime()
                << " ms" << std::endl
              << "***    Copy Bandwidth: " 
                << 1e-9 * memCpyD2HTimer.getBandwidth(numElements * sizeof(h_particles))
                << " GB/s" << std::endl
              << "***    Time for n-Body Computation: " << 1e3 * kernelTimer.getTime()
                << " ms" << std::endl
		<< "***    Body-body interactions/s [G]: " <<  1e-9*numElements*(numElements-1)*numIterations/kernelTimer.getTime()
                << std::endl
              << "***" << std::endl;
	
	std::fstream fin,fout;
	const char* fname;
	if(!optimized)
	{
		fname = "nBody.txt";
	}
	else
	{
		fname = "nBody_opt.txt";
	}
	fout.open(fname, std::ios::in | std::ios::out | std::ios::app);
	fin.open(fname, std::ios::in);
	if (!fout)
	{
	std::cerr << "file open failed:" << std::endl; 
	}
	if(fin.peek() == std::fstream::traits_type::eof())
	{
		fout << "numElements" << " " << "blockDim" << " " << "bodyBodyInt[G]" << "\n";
	} 
		fout << numElements << " " << block_dim.x << " " << 1e-9*numElements*(numElements-1)*numIterations/kernelTimer.getTime() << "\n"; 
		
	fin.close();
	fout.close();
	return 0;
}

void
printHelp(char * argv)
{
    std::cout << "Help:" << std::endl
              << "  Usage: " << std::endl
              << "  " << argv << " [-p] [-s <num-elements>] [-t <threads_per_block>]"
                  << std::endl
              << "" << std::endl
              << "  -p|--pinned-memory" << std::endl
              << "    Use pinned Memory instead of pageable memory" << std::endl
              << "" << std::endl
              << "  -s <num-elements>|--size <num-elements>" << std::endl
              << "    Number of elements (particles)" << std::endl
              << "" << std::endl
              << "  -t <threads_per_block>|--threads-per-block <threads_per_block>" 
                  << std::endl
              << "    The number of threads per block" << std::endl
              << "" << std::endl;
}

//
// Print one element
//
void
printElement(Body_t particles, int elementId, int iteration)
{
    float4 posMass = particles.posMass[elementId];
    float3 velocity = particles.velocity[elementId];

    std::cout << "***" << std::endl
              << "*** Printing Element " << elementId << " in iteration " << iteration << std::endl
              << "***" << std::endl
              << "*** Position: <" 
                  << std::setw(11) << std::setprecision(9) << posMass.x << "|"
                  << std::setw(11) << std::setprecision(9) << posMass.y << "|"
                  << std::setw(11) << std::setprecision(9) << posMass.z << "> [m]" << std::endl
              << "*** velocity: <" 
                  << std::setw(11) << std::setprecision(9) << velocity.x << "|"
                  << std::setw(11) << std::setprecision(9) << velocity.y << "|"
                  << std::setw(11) << std::setprecision(9) << velocity.z << "> [m/s]" << std::endl
              << "*** Mass: " 
                  << std::setw(11) << std::setprecision(9) << posMass.w << " kg"<< std::endl
              << "***" << std::endl;
}
