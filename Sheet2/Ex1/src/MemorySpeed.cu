#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"

__global__
void
NullKernel()
{
}

int
main(int argc,char** argv)
{
    const int cIterations = 1000;
    printf( "Measuring memory transfer... " ); //fflush( stdout );
 
    chTimerTimestamp start, stop;
    size_t freemem, totalmem;

    int N = atoi(argv[1]);
    printf("\n Size: %li \n",N*sizeof(float));
    void *dmem;
    hipMalloc (&dmem,N*sizeof ( float ) ); // Allocate GPU memory 
    void *hmem;
    hmem = (void *) malloc ( N*sizeof ( float ) );     // Allocate CPU memory 
    hipMemGetInfo(&freemem,&totalmem);

    printf("%li KB free of total %li KB\n",freemem/1024,totalmem/1024); 
    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
    // Transfer data from host to device 
    hipMemcpy ( dmem, hmem, N*sizeof ( float ), hipMemcpyHostToDevice ); 
    //hipMemcpy ( hmem, dmem, N*sizeof ( float ), hipMemcpyDeviceToHost ); 

    }   
 
    //free ( hmem );      // Free host buffer
    hipDeviceSynchronize();
    chTimerGetTime( &stop );
    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f us\n", usPerLaunch );
	FILE *fp;
	fp = fopen("MemoryMallocH2D.txt","a");
	fprintf (fp, "%li %.5g\n", N*sizeof(float), usPerLaunch );
	fclose(fp);
    }
    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
    // Transfer data from host to device 
    //hipMemcpy ( dmem, hmem, N*sizeof ( float ), hipMemcpyHostToDevice ); 
    hipMemcpy ( hmem, dmem, N*sizeof ( float ), hipMemcpyDeviceToHost ); 

    }   
 
    //free ( hmem );      // Free host buffer
    hipDeviceSynchronize();
    chTimerGetTime( &stop );
    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f us\n", usPerLaunch );
	FILE *fp;
	fp = fopen("MemoryMallocD2H.txt","a");
	fprintf (fp, "%li %.5g\n", N*sizeof(float), usPerLaunch );
	fclose(fp);
    }
    free(hmem);
    hipHostMalloc(&hmem,N*sizeof ( float ) );

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
    // Transfer data from host to device 
    hipMemcpy ( dmem, hmem, N*sizeof ( float ), hipMemcpyHostToDevice ); 
    //hipMemcpy ( hmem, dmem, N*sizeof ( float ), hipMemcpyDeviceToHost ); 

    }   
 
    //free ( hmem );      // Free host buffer
    hipDeviceSynchronize();
    chTimerGetTime( &stop );
    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f us\n", usPerLaunch );
	FILE *fp;
	fp = fopen("MemoryMallocHostH2D.txt","a");
	fprintf (fp, "%li %.5g\n", N*sizeof(float), usPerLaunch );
	fclose(fp);
    }
    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
    // Transfer data from host to device 
    //hipMemcpy ( dmem, hmem, N*sizeof ( float ), hipMemcpyHostToDevice ); 
    hipMemcpy ( hmem, dmem, N*sizeof ( float ), hipMemcpyDeviceToHost ); 

    }   
 
    //free ( hmem );      // Free host buffer
    hipDeviceSynchronize();
    chTimerGetTime( &stop );

    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f us\n", usPerLaunch );
	FILE *fp;
	fp = fopen("MemoryMallocHostD2H.txt","a");
	fprintf (fp, "%li %.5g\n", N*sizeof(float), usPerLaunch );
	fclose(fp);
    }
    hipFree ( dmem );  // Free device buffer
    hipFree(hmem);



    return 0;
}
