/******************************************************************************
 *
 *Computer Engineering Group, Heidelberg University - GPU Computing Exercise 04
 *
 *                  Group : TBD
 *
 *                   File : kernel.cu
 *
 *                Purpose : Memory Operations Benchmark
 *
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
//
// Test Kernel
//

__global__ void 
globalMem2SharedMem
(int a_size, float* g_array)
{
	int id = threadIdx.x;
	extern __shared__ float s_array[];
	int size = floorf(a_size/sizeof(float));
	for (int i = 1;i*blockDim.x <= size; i++)
	{
		//printf("upper:%d\n",id);
		s_array[id] = g_array[id];
		id  += blockDim.x;
		__syncthreads();
	}
	if (id < size)
	{
		//printf("lower:%d\n",id);
		s_array[id] = g_array[id];
	}
	if (threadIdx.x == 0)
		g_array[0] = 1.0;
}

void globalMem2SharedMem_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, float* g_array) {
	globalMem2SharedMem<<< gridSize, blockSize, shmSize >>>(shmSize, g_array);
}

__global__ void 
SharedMem2globalMem
(int a_size,float* g_array)
{
	int id = threadIdx.x;
	extern __shared__ float s_array[];
	int size = floorf(a_size/sizeof(float));
	for (int i = 1;i*blockDim.x <= size; i++)
	{
		//printf("upper:%d\n",id);
		g_array[id] = s_array[id];
		id  += blockDim.x;
		__syncthreads();
	}
	if (id < size)
	{
		//printf("lower:%d\n",id);
		g_array[id] = s_array[id];
	}
	//if (threadIdx.x == 0) //PROBABLY NOT NEEDED
		//g_array[0] = 1.0;
}
void SharedMem2globalMem_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize,float* g_array) {
	SharedMem2globalMem<<< gridSize, blockSize, shmSize >>>(shmSize,g_array);
}

__global__ void 
SharedMem2Registers
//(/*TODO Parameters*/)
(int a_size, float* g_array)
{
	int id = threadIdx.x;
	float reg;  // single float should always be stored in register I think
	extern __shared__ float s_array[];
	int size = floorf(a_size/sizeof(float));
	for (int i = 1;i*blockDim.x <= size; i++)
	{
		//printf("upper:%d\n",id);
		reg = s_array[id];
		id  += blockDim.x;
		__syncthreads();
	}
	if (id < size)
	{
		//printf("lower:%d\n",id);
		reg = s_array[id];
	}
	if (threadIdx.x == 0)
		g_array[0] = 1.0;
}
void SharedMem2Registers_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, float* g_array) {
	SharedMem2Registers<<< gridSize, blockSize, shmSize >>>(shmSize,g_array);
}

__global__ void 
Registers2SharedMem
//(/*TODO Parameters*/)
(int a_size, float* g_array)
{
	int id = threadIdx.x;
	float reg = 4.0;  // single float should always be stored in register I think
	extern __shared__ float s_array[];
	int size = floorf(a_size/sizeof(float));
	for (int i = 1;i*blockDim.x <= size; i++)
	{
		//printf("upper:%d\n",id);
		s_array[id] = reg;
		id  += blockDim.x;
		__syncthreads();
	}
	if (id < size)
	{
		//printf("lower:%d\n",id);
		s_array[id] = reg;
	}
	if (threadIdx.x == 0)
		g_array[0] = 1.0;
}
void Registers2SharedMem_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, float* g_array) {
	Registers2SharedMem<<< gridSize, blockSize, shmSize >>>(shmSize, g_array);
}

__global__ void 
bankConflictsRead
//(/*TODO Parameters*/)
(int stride,long int* dClocks, float* g_array)
{
	long long start,end;
	int id = stride*threadIdx.x;
	float reg = 4.0;  // single float should always be stored in register I think
	extern __shared__ float s_array[];

	start = clock64();
	for(int i = 0;i<100;i++)
	{
		__syncthreads();
		reg = s_array[id];
	}
	end = clock64();
	long long time = end - start;
	*dClocks += time;
	if (threadIdx.x == 0)
		g_array[0] = reg;
}

void bankConflictsRead_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, int stride,long int* dClocks,float* g_array) {
	bankConflictsRead<<< gridSize, blockSize, shmSize >>>(stride,dClocks,g_array);
	hipDeviceSynchronize();
}
