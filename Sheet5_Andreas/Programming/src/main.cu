#include "hip/hip_runtime.h"
/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing ExercthreadXe 05
 *
 *                                 Group : TODO
 *
 *                                  File : main.cu
 *
 *                               Purpose : Naive Matrix Multiplication
 *
 *************************************************************************************************/

#include <cmath>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <chCommandLine.h>
#include <chTimer.hpp>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "mmult_cpu.h"

const static int DEFAULT_MATRIX_WIDTH = 1024;
const static int DEFAULT_BLOCK_DIM = 32;

//
// Function Prototypes
//
void printHelp(char * /*programName*/);

/**
* Naive Kernel implementation without shared memory: 5.2
* C = A*B
**/
__global__ void
matMul_Kernel(int matrixSize, float* matrixA, float* matrixB, float* matrixC)
{
    // Jeder Thread berechnet einen Value der Ergebnis Matrix
    // Column
    int elementIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // Row
    int elementIdy = blockIdx.y * blockDim.y + threadIdx.y;

    int elementId = elementIdy * matrixSize + elementIdx;
    // The result value the thread calculates
    float c_res = 0;
    // As usual: Check index does not exceed matrix dimensions
    if (elementIdx < matrixSize && elementIdy < matrixSize) {
        for(int i = 0; i < matrixSize; ++i){
            // Calculate the required index
            c_res += matrixA[elementIdy * matrixSize + i] * matrixB[i * matrixSize + elementIdx];
        }
        // We can write c_res to the result matrix position of the thread
        // Remember: One thread calcs one result of the matrix
        matrixC[elementIdy * matrixSize + elementIdx] = c_res;
    }
}

//
// Shared matMul_Kernel
//
__global__ void
shMatMul_Kernel(int matrixSize, float *matrixA, float *matrixB, float *matrixC)
{

    // HEIGHT = WIDTH
    int TILEWIDTH = blockDim.x;
    int TILEHEIGHT = blockDim.y;
    extern __shared__ float sh_Mem[];
    float *sh_MatrixA = &(sh_Mem[0]);
    float *sh_MatrixB = &(sh_Mem[TILEWIDTH * TILEHEIGHT]);
    // Why would we need shared mem for C?
    // float *sh_MatrixC = &(sh_Mem[2 /*TODO Calc offset*/]);

    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;
    int row = blockY * TILEHEIGHT + threadY;
    int col = blockX * TILEWIDTH + threadX;

    float Cvalue = 0.0;
    // Iterate the tiles, loop the submatrices
    for (int m = 0; m < gridDim.x; ++m)
    {
        // Load A and B from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        // (row, col) = row * width + col
        // A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col

        // If the matrix width is not a multiple of tile dim, we have to zero-out
        if (m * TILEWIDTH + threadX < matrixSize && row < matrixSize)
        {
            sh_MatrixA[threadY * TILEWIDTH + threadX] = matrixA[row * matrixSize + (m * TILEWIDTH + threadX)];
        }
        else
        {
            sh_MatrixA[threadY * TILEWIDTH + threadX] = 0.0;
        }
        if (m * TILEWIDTH + threadY < matrixSize && col < matrixSize)
        {
            sh_MatrixB[threadY * TILEWIDTH + threadX] = matrixB[col + (m * TILEWIDTH + threadY) * matrixSize];
        }
        else
        {
            sh_MatrixB[threadY * TILEWIDTH + threadX] = 0.0;
        }
        // Wait for operation to complete
        __syncthreads();

        // Multiply the sub matrices
        for (int e = 0; e < TILEWIDTH; ++e)
        {
            Cvalue += sh_MatrixA[threadY * TILEWIDTH + e] * sh_MatrixB[e * TILEWIDTH + threadX];
        }

        // Lets wait for all to complete before continuing
        __syncthreads();
    }
    if (row < matrixSize &&
        col < matrixSize)
    {
        // Write real result back
        matrixC[row * matrixSize + col] = Cvalue;
    }
}

// Main
int main(int argc, char *argv[])
{
    //
    // Show Help
    //
    bool showHelp = chCommandLineGetBool("h", argc, argv);
    if (!showHelp)
    {
        showHelp = chCommandLineGetBool("help", argc, argv);
    }

    if (showHelp)
    {
        printHelp(argv[0]);
        exit(0);
    }
    std::cout << "**************************************************************************\n";
    std::cout << "***" << std::endl
              << "*** Starting ..." << std::endl
              << "***" << std::endl;

    ChTimer memCpyH2DTimer, memCpyD2HTimer;
    ChTimer kernelTimer, CPUTimer;

    // More would be better, but it just takes too long
    const int ITERATIONS = 10;
    //
    // Allocate Memory
    //
    int matrixWidth = 0;
    chCommandLineGet<int>(&matrixWidth, "s", argc, argv);
    chCommandLineGet<int>(&matrixWidth, "size", argc, argv);
    matrixWidth = matrixWidth != 0 ? matrixWidth : DEFAULT_MATRIX_WIDTH;

    int matrixSize = matrixWidth * matrixWidth;

    //
    // Host Memory
    //
    bool pinnedMemory = chCommandLineGetBool("p", argc, argv);
    if (!pinnedMemory)
    {
        pinnedMemory = chCommandLineGetBool("pinned-memory", argc, argv);
    }

    float *h_matrixA = NULL;
    float *h_matrixB = NULL;
    float *h_matrixC = NULL;
    if (!pinnedMemory)
    {
        // Pageable
        h_matrixA = static_cast<float *>(malloc(
            static_cast<size_t>(matrixSize * sizeof(*h_matrixA))));
        h_matrixB = static_cast<float *>(malloc(
            static_cast<size_t>(matrixSize * sizeof(*h_matrixB))));
        h_matrixC = static_cast<float *>(calloc(
            static_cast<size_t>(matrixSize), sizeof *h_matrixC));
    }
    else
    {
        // Pinned
        hipHostMalloc(&h_matrixA, static_cast<size_t>(matrixSize * sizeof(*h_matrixA)));
        hipHostMalloc(&h_matrixB, static_cast<size_t>(matrixSize * sizeof(*h_matrixB)));
        hipHostMalloc(&h_matrixC, static_cast<size_t>(matrixSize * sizeof(*h_matrixC)));
        memset(h_matrixC, 0, matrixSize * sizeof(*h_matrixC));
    }

    //
    // Device Memory
    //
    float *d_matrixA = NULL;
    float *d_matrixB = NULL;
    float *d_matrixC = NULL;
    hipMalloc(&d_matrixA, static_cast<size_t>(matrixSize * sizeof(*d_matrixA)));
    hipMalloc(&d_matrixB, static_cast<size_t>(matrixSize * sizeof(*d_matrixB)));
    hipMalloc(&d_matrixC, static_cast<size_t>(matrixSize * sizeof(*d_matrixC)));

    //
    // Check Pointers
    //
    if (h_matrixA == NULL || h_matrixB == NULL || h_matrixC == NULL ||
        d_matrixA == NULL || d_matrixB == NULL || d_matrixC == NULL)
    {
        std::cout << "\033[31m***" << std::endl
                  << "*** Error - Allocation of Memory failed!!!" << std::endl
                  << "***\033[0m" << std::endl;
        exit(-1);
    }

    //
    // Init Matrices
    //
    for (int i = 0; i < matrixSize; i++)
    {
        int x = i % matrixWidth;
        int y = i / matrixWidth;
        h_matrixA[i] = static_cast<float>(x * y);
        h_matrixB[i] = static_cast<float>(x + y);
    }

    //
    // Copy Data to the Device
    //
    memCpyH2DTimer.start();

    hipMemcpy(d_matrixA, h_matrixA, static_cast<size_t>(matrixSize * sizeof(*d_matrixA)),
               hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, h_matrixB, static_cast<size_t>(matrixSize * sizeof(*d_matrixB)),
               hipMemcpyHostToDevice);

    memCpyH2DTimer.stop();

    //
    // Get Kernel Launch Parameters
    //
    int blockSize = 0,
        gridSize = 0;

    // Block Dimension / Threads per Block
    chCommandLineGet<int>(&blockSize, "t", argc, argv);
    chCommandLineGet<int>(&blockSize, "threads-per-block", argc, argv);
    blockSize = blockSize != 0 ? blockSize : DEFAULT_BLOCK_DIM;

    if (blockSize > 32)
    {
        std::cout << "\033[31m***" << std::endl
                  << "*** Error - The number of threads per block is too high" << std::endl
                  << "***\033[0m" << std::endl;
        exit(-1);
    }

    gridSize = ceil(static_cast<float>(matrixWidth) / static_cast<float>(blockSize));

    dim3 grid_dim = dim3(gridSize, gridSize, 1);
    dim3 block_dim = dim3(blockSize, blockSize, 1);

    std::cout << "***" << std::endl
              << "*** Grid Dim:  " << grid_dim.x << "x" << grid_dim.y << "x" << grid_dim.z
              << std::endl
              << "*** Block Dim: " << block_dim.x << "x" << block_dim.y << "x" << block_dim.z
              << std::endl
              << "***" << std::endl;

    // TODO Calc shared mem size
    int sharedMemSize = 2 * pow(blockSize, 2) * sizeof(float);
    kernelTimer.start();

    //
    // Launch Kernel for i ITERATIONS
    //
    for (int i = 0; i < ITERATIONS; ++i)
    {
        if (!chCommandLineGetBool("shared", argc, argv))
        {
            matMul_Kernel<<<grid_dim, block_dim>>>(matrixWidth, d_matrixA, d_matrixB, d_matrixC);
        }
        else
        {
            shMatMul_Kernel<<<grid_dim, block_dim, sharedMemSize>>>(matrixWidth, d_matrixA, d_matrixB, d_matrixC);
        }

        //
        // Synchronize
        //
        hipDeviceSynchronize();
    }
    //
    // Check for Errors
    //
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        std::cout << "\033[31m***" << std::endl
                  << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
                  << std::endl
                  << "***\033[0m" << std::endl;

        return -1;
    }

    kernelTimer.stop();

    if (!chCommandLineGetBool("shared", argc, argv))
    {
        std::cout << "Using matMul_Kernel\n";
    }
    else
    {
        std::cout << "Using shMatMul_Kernel\n";
    }
    //
    // Copy Back Data
    //
    memCpyD2HTimer.start();

    hipMemcpy(h_matrixC, d_matrixC, static_cast<size_t>(matrixSize * sizeof(*d_matrixC)),
               hipMemcpyDeviceToHost);

    memCpyD2HTimer.stop();

    //
    // Check Result
    //
    bool dontCheckResult = chCommandLineGetBool("c", argc, argv);
    if (!dontCheckResult)
    {
        dontCheckResult = chCommandLineGetBool("no-check", argc, argv);
    }

    float cpu_mat_time = 0;
    if (!dontCheckResult)
    {
        std::cout << "Now checking results...\n";
        float *h_matrixD = static_cast<float *>(
            calloc(static_cast<size_t>(matrixSize), sizeof(*h_matrixD)));

        CPUTimer.start();
        MatrixMulOnHostBlocked(h_matrixA, h_matrixB, h_matrixD,
                               static_cast<long>(matrixWidth), 32);
        CPUTimer.stop();
        std::cout << "CPU took " << 1e3 * CPUTimer.getTime() << "ms." << std::endl;
        cpu_mat_time = 1e3 * CPUTimer.getTime();
        bool resultOk = MatrixCompare(h_matrixC, h_matrixD,
                                      static_cast<long>(matrixWidth));

        if (!resultOk)
        {
            std::cout << "\033[31m***" << std::endl
                      << "*** Error - The two matrices are different!!!" << std::endl
                      << "***\033[0m" << std::endl;
            printOutMatrix(h_matrixA, matrixWidth);
            printOutMatrix(h_matrixB, matrixWidth);
            printOutMatrix(h_matrixC, matrixWidth);
            printOutMatrix(h_matrixD, matrixWidth);

            exit(-1);
        }
        else
        {
            std::cout << "The two matrices are the same\n";
        }

        free(h_matrixD);
    }
    else
    {
        std::cout << "Result CPU Check disabled\n";
    }

    //
    // Print Meassurement Results
    //
    std::cout << "***" << std::endl
              << "*** Results:" << std::endl
              << "*** Matrix width:" << matrixWidth << std::endl
              << "***    Matrix Size (width*width): " << matrixSize << std::endl
              << "***    Grid size: " << gridSize << std::endl
              << "***    Block size : " << blockSize << std::endl
              << "***    Time to Copy to Device: " << 1e3 * memCpyH2DTimer.getTime()
              << " ms" << std::endl
              << "***    Copy Bandwidth: "
              << 1e-9 * memCpyH2DTimer.getBandwidth(2 * matrixSize * sizeof(*h_matrixA))
              << " GB/s" << std::endl
              << "***    Time to Copy from Device: " << 1e3 * memCpyD2HTimer.getTime()
              << " ms" << std::endl
              << "***    Copy Bandwidth: "
              << 1e-9 * memCpyD2HTimer.getBandwidth(matrixSize * sizeof(*h_matrixA))
              << " GB/s" << std::endl
              << "***    Time for Matrix Multiplication: " << 1e3 * kernelTimer.getTime() / ITERATIONS
              << " ms" << std::endl
              << "***" << std::endl;
    // Print csv style for plotting
    std::cout << "CSV\n matrixWidth, blockSize, H2D_ms, D2H_ms, matrixMult_ms_complCalc, matrixMult_ms, ITERATIONS, CPU Time\n";
    std::cout << matrixWidth << ", " << blockSize << ", " << 1e3 * memCpyH2DTimer.getTime() << ", " << 1e3 * memCpyD2HTimer.getTime() << ", " << 1e3 * kernelTimer.getTime() << ", " << 1e3 * kernelTimer.getTime() / ITERATIONS << ", " << ITERATIONS << ", " << cpu_mat_time << std::endl;
    std::cout << "CSV\n";
    std::cout << "**************************************************************************\n";
    if (chCommandLineGetBool("print-matrix", argc, argv) && matrixWidth <= 16)
    {
        printOutMatrix(h_matrixC, matrixWidth);
    }

    // Free Memory
    if (!pinnedMemory)
    {
        free(h_matrixA);
        free(h_matrixB);
        free(h_matrixC);
    }
    else
    {
        hipHostFree(h_matrixA);
        hipHostFree(h_matrixB);
        hipHostFree(h_matrixC);
    }
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);

    return 0;
}

void printHelp(char *programName)
{
    std::cout << "Help:" << std::endl
              << "  Usage: " << std::endl
              << "  " << programName << " [-p] [-s <matrix_size>] [-t <threads_per_block>]"
              << std::endl
              << "                 [-g <blocks_per_grid] [-c] [--print-matrix]"
              << std::endl
              << "" << std::endl
              << "  -p|--pinned-memory" << std::endl
              << "  Use pinned Memory instead of pageable memory" << std::endl
              << "" << std::endl
              << "  -s <matrix_size>|--size <matix_size>" << std::endl
              << "  The width of the Matrix" << std::endl
              << "" << std::endl
              << "  -t <threads_per_block>|--threads-per-block <threads_per_block>"
              << std::endl
              << "  The number of threads per block" << std::endl
              << "" << std::endl
              << "  -c|--no-checking" << std::endl
              << "  Do not check the result of the matrix multiplication" << std::endl
              << "" << std::endl
              << "  --print-matrix" << std::endl
              << "  Print the output matrix (only recommended for small matrices)" << std::endl
              << std::endl;
}
