#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"
__global__
void
Init()
{
}

int
main(int argc,char** argv)
{
    (void) argc; //so compiler stops being annoying
    const int cIterations = 100000000;
    printf( "Measuring memory transfer... " ); //fflush( stdout );
 
    chTimerTimestamp start, stop;

    long int size = atoi(argv[1])*sizeof(char);
    char *dmem = NULL; hipMalloc((void**) dmem,size);
    char *dmem2 = NULL; hipMalloc((void**) dmem2,size);
    char *hmem = (char*) malloc(size);
    for (int i = 0; i<size; i++)
    {
    hmem[i] = 's';
    }
    hipMemcpy(hmem,dmem,size,hipMemcpyHostToDevice);
    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
    hipMemcpy ( dmem, dmem2, size, hipMemcpyDeviceToDevice ); 

    }   
 
    hipDeviceSynchronize();
    chTimerGetTime( &stop );
    {
        double bandwidth = (size*cIterations)/chTimerElapsedTime( &start, &stop );
	printf( "Message Size: %li Byte\n", size );
        printf( "Bandwidth: %.5g Byte/s\n", bandwidth );
	FILE *fp;
	fp = fopen("MemorySpeedD2D.txt","a");
	fprintf (fp, "%li %.5g\n", size, bandwidth );
	fclose(fp);
    }

    hipFree(dmem);  // Free device buffer
    hipFree(dmem2);



    return 0;
}
